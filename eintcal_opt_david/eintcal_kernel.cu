#include "hip/hip_runtime.h"
// includes, system
//#include <hip/hip_runtime.h>
//#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
//#include "typedefs.h"

//C++ #defines
#define USE_8A_NBCUTOFF
#include "autocomm.h"
//#include "grid.h"
//#include "eval.h"
#include "constants.h"
//#include "trilinterp.h"
//#include "eintcal.h"
//#include "distdepdiel.h"
#include "cuda_wrapper.h"

// Dummy input
#include "dummyinput.h"

// other defines
#define NOSQRT
#define BLOCK_SIZE 128

// Constant cache
__constant__ float nonbondlist_c[NONBONDLISTS_SIZE];
__constant__ int nnb_array_c[NNB_ARRAY_SIZE];
__constant__ float strsol_fn_c[SOLFN_SIZE];

/**
 * eintcal GPU kernel, does eintcal energy calculations for each 
 * individual in the population.
 * @param num_individualsgpu number of individuals in population
 * @param natomsgpu number of atoms
 * @param penergiesgpu array of energies used to store individual's energy
 * @param nonbondlist (used in cpu eintcal)
 * @param tcoord (used in cpu eintcal)
 * @param B_include_1_4_interactions (used in cpu eintcal)
 * @param B_have_flexible_residues (used in cpu eintcal)
 * @param nnb_array (used in cpu eintcal)
 * @param Nb_group_energy (used in cpu eintcal)
 * @param stre_vdW_Hb (used in cpu eintcal)
 * @param strsol_fn (used in cpu eintcal)
 * @param strepsilon_fn (used in cpu eintcal)
 * @param strr_epsilon_fn (used in cpu eintcal)
 * @param b_comp_intermolgpu (used in cpu eintcal)
 * @param pfloat_arraygpu array of float variables used in cpu trilinterp
 * @param pint_arraygpu array of integer varibales used in cpu trilinterp
 */
__global__ void eintcal_kernel(unsigned int num_individualsgpu,
								int natomsgpu, 
								float *penergiesgpu, 
								float *nonbondlist, 
								float *tcoord, 
								int B_include_1_4_interactions, // Boole
								int B_have_flexible_residues, // Boole
								int *nnb_array, 
								float *Nb_group_energy, 
								float *stre_vdW_Hb, 
								float *strsol_fn, 
								float *strepsilon_fn, 
								float *strr_epsilon_fn,
								int b_comp_intermolgpu, // Boole
								float *pfloat_arraygpu,
								int *pint_arraygpu)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	

	if (idx < num_individualsgpu) {
		
		if (!pint_arraygpu[INTEVALFLAG * num_individualsgpu + idx])//!evalflagsgpu[idx])
		{
			
# ifndef NOSQRT
			float r = 0.0f;
			// float nbc = B_use_non_bond_cutoff[idx] ? NBC : 999;
			float nbc = (int)pint_arraygpu[INTNONBONDCUT * num_individualsgpu + idx] ? NBC : 999; // Boole cast
# else
			// float nbc2 = B_use_non_bond_cutoff[idx] ? NBC2 : 999 * 999;
			float nbc2 = (int)pint_arraygpu[INTNONBONDCUT * num_individualsgpu + idx] ? NBC2 : 999 * 999; // Boole cast
# endif
			float dx = 0.0f, dy = 0.0f, dz = 0.0f;
			float r2 = 0.0f;
			
			float total_e_internal = 0.0f;
			
			float e_elec = 0.0f;
			
			int inb = 0;
			int a1 = 0, a2 = 0;
			int t1 = 0, t2 = 0;
			int nonbond_type = 0;
			
			int index_1t_NEINT = 0;
			int index_1t_NDIEL = 0;
			int nb_group = 0;
			int inb_from = 0;
			int inb_to = 0;
			int nb_group_max = 1;
			
			if (B_have_flexible_residues) {
				nb_group_max = 3;
			}
			
			for (nb_group = 0; nb_group < nb_group_max; nb_group++)
			{
				if (nb_group == 0) {
					inb_from = 0;
				} else {
					inb_from = nnb_array[nb_group-1];
				}
				inb_to = nnb_array[nb_group];
				
				for (inb = inb_from; inb < inb_to; inb++)
				{
				
					float e_internal = 0.0f;
					float e_desolv = 0.0f;
					
					a1 = (int)nonbondlist[inb * 7 + 0];
					a2 = (int)nonbondlist[inb * 7 + 1];
					t1 = (int)nonbondlist[inb * 7 + 2];
					t2 = (int)nonbondlist[inb * 7 + 3];
					
					nonbond_type = (int)nonbondlist[inb * 7 + 4];
					float nb_desolv = nonbondlist[inb * 7 + 5];
					float q1q2 = nonbondlist[inb * 7 + 6];
					
					dx = tcoord[idx * natomsgpu * SPACE + a1 * SPACE + X] - tcoord[idx * natomsgpu * SPACE + a2 * SPACE + X];
					dy = tcoord[idx * natomsgpu * SPACE + a1 * SPACE + Y] - tcoord[idx * natomsgpu * SPACE + a2 * SPACE + Y];
					dz = tcoord[idx * natomsgpu * SPACE + a1 * SPACE + Z] - tcoord[idx * natomsgpu * SPACE + a2 * SPACE + Z];
					
	#ifndef NOSQRT
					r = clamp(hypotenuse(dx,dy,dz), RMIN_ELEC);
					r2 = r*r;
					int index = Ang_to_index(r);
					
	#else
					r2 = sqhypotenuse(dx,dy,dz);
					r2 = clamp(r2, RMIN_ELEC2);
					int index = SqAng_to_index(r2);
	#endif
					
					index_1t_NEINT = BoundedNeint(index);
					index_1t_NDIEL = BoundedNdiel(index);
					
					if ((int)pint_arraygpu[INTINCELEC * num_individualsgpu + idx])//B_calcIntElec[idx]) // Boole cast
					{
						float r_dielectric = strr_epsilon_fn[index_1t_NDIEL];
						e_elec = q1q2 * r_dielectric;
						e_internal = e_elec;
					}
					
					if (r2 < nbc2) {
						e_desolv = strsol_fn[index_1t_NEINT] * nb_desolv;
						int myidx;
						if (B_include_1_4_interactions != 0 && nonbond_type == 4) {
							myidx = index_1t_NEINT * ATOM_MAPS * ATOM_MAPS + t2 * ATOM_MAPS + t1;
							if (myidx == NEINT * ATOM_MAPS * ATOM_MAPS) {
								// e_internal += scale_1_4[idx] * (stre_vdW_Hb[myidx-1] + e_desolv);
								e_internal += pfloat_arraygpu[FLOATSCALE14 * num_individualsgpu + idx] * (stre_vdW_Hb[myidx-1] + e_desolv);
							}
							else {
								// e_internal += scale_1_4[idx] * (stre_vdW_Hb[myidx] + e_desolv);
								e_internal += pfloat_arraygpu[FLOATSCALE14 * num_individualsgpu + idx] * (stre_vdW_Hb[myidx] + e_desolv);
							}
						} else {
							// fprintf(stderr," stre_vdW_Hb[%d][%d][%d] = %f\n", index_1t_NEINT, t2, t1, stre_vdW_Hb[index_1t_NEINT * ATOM_MAPS * ATOM_MAPS + t2 * ATOM_MAPS + t1]);
							// e_internal += stre_vdW_Hb[index_1t_NEINT * ATOM_MAPS * ATOM_MAPS + t2 * ATOM_MAPS + t1] + e_desolv;
							myidx = index_1t_NEINT * ATOM_MAPS * ATOM_MAPS + t2 * ATOM_MAPS + t1;
							if (myidx == NEINT * ATOM_MAPS * ATOM_MAPS) {
								e_internal += stre_vdW_Hb[myidx-1] + e_desolv;
								// fprintf(stderr,"NEINT = %d, index = %d, t2 = %d, t1 = %d\n", NEINT, index_1t_NEINT, t2, t1);
							}
							else {
								e_internal += stre_vdW_Hb[myidx] + e_desolv;
							}
					
						}
						
						
						
					}
					total_e_internal += e_internal;
				}
				
				if (nb_group == INTRA_LIGAND) 
				{
					Nb_group_energy[INTRA_LIGAND] = total_e_internal;
				} else if (nb_group == INTER) {
					Nb_group_energy[INTER] = total_e_internal - Nb_group_energy[INTRA_LIGAND];
				} else if (nb_group == INTRA_RECEPTOR) {
					Nb_group_energy[INTRA_RECEPTOR] = total_e_internal - Nb_group_energy[INTRA_LIGAND] - Nb_group_energy[INTER];
				}
			}
			
			if(b_comp_intermolgpu) {
				//energiesgpu[idx] += ((float)total_e_internal - (float)unboundinternalFEs[idx]);
				penergiesgpu[idx] += ((float)total_e_internal - pfloat_arraygpu[FLOATUNBOUNDINTERNAL * num_individualsgpu + idx]);//(float)unboundinternalFEs[idx]);
			}
			else {
				//energiesgpu[idx] = ((float)total_e_internal - (float)unboundinternalFEs[idx]);
				penergiesgpu[idx] = ((float)total_e_internal - pfloat_arraygpu[FLOATUNBOUNDINTERNAL * num_individualsgpu + idx]);//(float)unboundinternalFEs[idx]);
			}
		}
	}
}

__global__ void eintcal_kernel_shared(unsigned int num_individualsgpu,
								int natomsgpu, 
								float *penergiesgpu, 
								float *nonbondlist, 
								float *tcoord, 
								int B_include_1_4_interactions, // Boole
								int B_have_flexible_residues, // Boole
								int *nnb_array, 
								float *Nb_group_energy, 
								float *stre_vdW_Hb, 
								float *strsol_fn, 
								float *strepsilon_fn, 
								float *strr_epsilon_fn,
								int b_comp_intermolgpu, // Boole
								float *pfloat_arraygpu,
								int *pint_arraygpu)
{
	__shared__ float nonbondlist_s[NONBONDLISTS_SIZE];
	for (int i = threadIdx.x; i < NONBONDLISTS_SIZE; i += blockDim.x) {
		nonbondlist_s[i] = nonbondlist[i];
	}

	__syncthreads();

	//int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = threadIdx.x;

	if (idx + blockDim.x * blockIdx.x < num_individualsgpu) {
		
		if (!pint_arraygpu[INTEVALFLAG * num_individualsgpu + idx])//!evalflagsgpu[idx])
		{
			
# ifndef NOSQRT
			float r = 0.0f;
			// float nbc = B_use_non_bond_cutoff[idx] ? NBC : 999;
			float nbc = (int)pint_arraygpu[INTNONBONDCUT * num_individualsgpu + idx] ? NBC : 999; // Boole cast
# else
			// float nbc2 = B_use_non_bond_cutoff[idx] ? NBC2 : 999 * 999;
			float nbc2 = (int)pint_arraygpu[INTNONBONDCUT * num_individualsgpu + idx] ? NBC2 : 999 * 999; // Boole cast
# endif
			float dx = 0.0f, dy = 0.0f, dz = 0.0f;
			float r2 = 0.0f;
			
			float total_e_internal = 0.0f;
			
			float e_elec = 0.0f;
			
			int inb = 0;
			int a1 = 0, a2 = 0;
			int t1 = 0, t2 = 0;
			int nonbond_type = 0;
			
			int index_1t_NEINT = 0;
			int index_1t_NDIEL = 0;
			int nb_group = 0;
			int inb_from = 0;
			int inb_to = 0;
			int nb_group_max = 1;
			
			if (B_have_flexible_residues) {
				nb_group_max = 3;
			}
			
			for (nb_group = 0; nb_group < nb_group_max; nb_group++)
			{
				if (nb_group == 0) {
					inb_from = 0;
				} else {
					inb_from = nnb_array[nb_group-1];
				}
				inb_to = nnb_array[nb_group];
				
				for (inb = inb_from; inb < inb_to; inb++)
				{
				
					float e_internal = 0.0f;
					float e_desolv = 0.0f;
					
					a1 = (int)nonbondlist_s[inb * 7 + 0];
					a2 = (int)nonbondlist_s[inb * 7 + 1];
					t1 = (int)nonbondlist_s[inb * 7 + 2];
					t2 = (int)nonbondlist_s[inb * 7 + 3];
					
					nonbond_type = (int)nonbondlist_s[inb * 7 + 4];
					float nb_desolv = nonbondlist_s[inb * 7 + 5];
					float q1q2 = nonbondlist_s[inb * 7 + 6];
					
					dx = tcoord[idx * natomsgpu * SPACE + a1 * SPACE + X] - tcoord[idx * natomsgpu * SPACE + a2 * SPACE + X];
					dy = tcoord[idx * natomsgpu * SPACE + a1 * SPACE + Y] - tcoord[idx * natomsgpu * SPACE + a2 * SPACE + Y];
					dz = tcoord[idx * natomsgpu * SPACE + a1 * SPACE + Z] - tcoord[idx * natomsgpu * SPACE + a2 * SPACE + Z];
					
	#ifndef NOSQRT
					r = clamp(hypotenuse(dx,dy,dz), RMIN_ELEC);
					r2 = r*r;
					int index = Ang_to_index(r);
					
	#else
					r2 = sqhypotenuse(dx,dy,dz);
					r2 = clamp(r2, RMIN_ELEC2);
					int index = SqAng_to_index(r2);
	#endif
					
					index_1t_NEINT = BoundedNeint(index);
					index_1t_NDIEL = BoundedNdiel(index);
					
					if ((int)pint_arraygpu[INTINCELEC * num_individualsgpu + idx])//B_calcIntElec[idx]) // Boole cast
					{
						float r_dielectric = strr_epsilon_fn[index_1t_NDIEL];
						e_elec = q1q2 * r_dielectric;
						e_internal = e_elec;
					}
					
					if (r2 < nbc2) {
						e_desolv = strsol_fn[index_1t_NEINT] * nb_desolv;
						int myidx;
						if (B_include_1_4_interactions != 0 && nonbond_type == 4) {
							myidx = index_1t_NEINT * ATOM_MAPS * ATOM_MAPS + t2 * ATOM_MAPS + t1;
							if (myidx == NEINT * ATOM_MAPS * ATOM_MAPS) {
								// e_internal += scale_1_4[idx] * (stre_vdW_Hb[myidx-1] + e_desolv);
								e_internal += pfloat_arraygpu[FLOATSCALE14 * num_individualsgpu + idx] * (stre_vdW_Hb[myidx-1] + e_desolv);
							}
							else {
								// e_internal += scale_1_4[idx] * (stre_vdW_Hb[myidx] + e_desolv);
								e_internal += pfloat_arraygpu[FLOATSCALE14 * num_individualsgpu + idx] * (stre_vdW_Hb[myidx] + e_desolv);
							}
						} else {
							// fprintf(stderr," stre_vdW_Hb[%d][%d][%d] = %f\n", index_1t_NEINT, t2, t1, stre_vdW_Hb[index_1t_NEINT * ATOM_MAPS * ATOM_MAPS + t2 * ATOM_MAPS + t1]);
							// e_internal += stre_vdW_Hb[index_1t_NEINT * ATOM_MAPS * ATOM_MAPS + t2 * ATOM_MAPS + t1] + e_desolv;
							myidx = index_1t_NEINT * ATOM_MAPS * ATOM_MAPS + t2 * ATOM_MAPS + t1;
							if (myidx == NEINT * ATOM_MAPS * ATOM_MAPS) {
								e_internal += stre_vdW_Hb[myidx-1] + e_desolv;
								// fprintf(stderr,"NEINT = %d, index = %d, t2 = %d, t1 = %d\n", NEINT, index_1t_NEINT, t2, t1);
							}
							else {
								e_internal += stre_vdW_Hb[myidx] + e_desolv;
							}
					
						}
						
						
					}
					total_e_internal += e_internal;
				}
				
				if (nb_group == INTRA_LIGAND) 
				{
					Nb_group_energy[INTRA_LIGAND] = total_e_internal;
				} else if (nb_group == INTER) {
					Nb_group_energy[INTER] = total_e_internal - Nb_group_energy[INTRA_LIGAND];
				} else if (nb_group == INTRA_RECEPTOR) {
					Nb_group_energy[INTRA_RECEPTOR] = total_e_internal - Nb_group_energy[INTRA_LIGAND] - Nb_group_energy[INTER];
				}
			}
			
			if(b_comp_intermolgpu) {
				//energiesgpu[idx] += ((float)total_e_internal - (float)unboundinternalFEs[idx]);
				penergiesgpu[idx] += ((float)total_e_internal - pfloat_arraygpu[FLOATUNBOUNDINTERNAL * num_individualsgpu + idx]);//(float)unboundinternalFEs[idx]);
			}
			else {
				//energiesgpu[idx] = ((float)total_e_internal - (float)unboundinternalFEs[idx]);
				penergiesgpu[idx] = ((float)total_e_internal - pfloat_arraygpu[FLOATUNBOUNDINTERNAL * num_individualsgpu + idx]);//(float)unboundinternalFEs[idx]);
			}
		}
	}
}

// const_1 uses more const memory arrays (nnb, nonbondlist, strsol)
__global__ void eintcal_kernel_const_1(unsigned int num_individualsgpu,
								int natomsgpu, 
								float *penergiesgpu, 
								float *tcoord, 
								int B_include_1_4_interactions, // Boole
								int B_have_flexible_residues, // Boole
								float *Nb_group_energy, 
								float *stre_vdW_Hb, 
								float *strepsilon_fn, 
								float *strr_epsilon_fn,
								int b_comp_intermolgpu, // Boole
								float *pfloat_arraygpu,
								int *pint_arraygpu)
{
	//int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = threadIdx.x;

	if (idx + blockDim.x * blockIdx.x < num_individualsgpu) {
		
		if (!pint_arraygpu[INTEVALFLAG * num_individualsgpu + idx])//!evalflagsgpu[idx])
		{
			
# ifndef NOSQRT
			float r = 0.0f;
			// float nbc = B_use_non_bond_cutoff[idx] ? NBC : 999;
			float nbc = (int)pint_arraygpu[INTNONBONDCUT * num_individualsgpu + idx] ? NBC : 999; // Boole cast
# else
			// float nbc2 = B_use_non_bond_cutoff[idx] ? NBC2 : 999 * 999;
			float nbc2 = (int)pint_arraygpu[INTNONBONDCUT * num_individualsgpu + idx] ? NBC2 : 999 * 999; // Boole cast
# endif
			float dx = 0.0f, dy = 0.0f, dz = 0.0f;
			float r2 = 0.0f;
			
			float total_e_internal = 0.0f;
			
			float e_elec = 0.0f;
			
			int inb = 0;
			int a1 = 0, a2 = 0;
			int t1 = 0, t2 = 0;
			int nonbond_type = 0;
			
			int index_1t_NEINT = 0;
			int index_1t_NDIEL = 0;
			int nb_group = 0;
			int inb_from = 0;
			int inb_to = 0;
			int nb_group_max = 1;
			
			if (B_have_flexible_residues) {
				nb_group_max = 3;
			}
			
			for (nb_group = 0; nb_group < nb_group_max; nb_group++)
			{
				if (nb_group == 0) {
					inb_from = 0;
				} else {
					inb_from = nnb_array_c[nb_group-1];
				}
				inb_to = nnb_array_c[nb_group];
				
				for (inb = inb_from; inb < inb_to; inb++)
				{
				
					float e_internal = 0.0f;
					float e_desolv = 0.0f;
					
					a1 = (int)nonbondlist_c[inb * 7 + 0];
					a2 = (int)nonbondlist_c[inb * 7 + 1];
					t1 = (int)nonbondlist_c[inb * 7 + 2];
					t2 = (int)nonbondlist_c[inb * 7 + 3];
					
					nonbond_type = (int)nonbondlist_c[inb * 7 + 4];
					float nb_desolv = nonbondlist_c[inb * 7 + 5];
					float q1q2 = nonbondlist_c[inb * 7 + 6];
					
					dx = tcoord[idx * natomsgpu * SPACE + a1 * SPACE + X] - tcoord[idx * natomsgpu * SPACE + a2 * SPACE + X];
					dy = tcoord[idx * natomsgpu * SPACE + a1 * SPACE + Y] - tcoord[idx * natomsgpu * SPACE + a2 * SPACE + Y];
					dz = tcoord[idx * natomsgpu * SPACE + a1 * SPACE + Z] - tcoord[idx * natomsgpu * SPACE + a2 * SPACE + Z];
					
	#ifndef NOSQRT
					r = clamp(hypotenuse(dx,dy,dz), RMIN_ELEC);
					r2 = r*r;
					int index = Ang_to_index(r);
					
	#else
					r2 = sqhypotenuse(dx,dy,dz);
					r2 = clamp(r2, RMIN_ELEC2);
					int index = SqAng_to_index(r2);
	#endif
					
					index_1t_NEINT = BoundedNeint(index);
					index_1t_NDIEL = BoundedNdiel(index);
					
					if ((int)pint_arraygpu[INTINCELEC * num_individualsgpu + idx])//B_calcIntElec[idx]) // Boole cast
					{
						float r_dielectric = strr_epsilon_fn[index_1t_NDIEL];
						e_elec = q1q2 * r_dielectric;
						e_internal = e_elec;
					}
					
					if (r2 < nbc2) {
						e_desolv = strsol_fn_c[index_1t_NEINT] * nb_desolv;
						int myidx;
						if (B_include_1_4_interactions != 0 && nonbond_type == 4) {
							myidx = index_1t_NEINT * ATOM_MAPS * ATOM_MAPS + t2 * ATOM_MAPS + t1;
							if (myidx == NEINT * ATOM_MAPS * ATOM_MAPS) {
								// e_internal += scale_1_4[idx] * (stre_vdW_Hb[myidx-1] + e_desolv);
								e_internal += pfloat_arraygpu[FLOATSCALE14 * num_individualsgpu + idx] * (stre_vdW_Hb[myidx-1] + e_desolv);
							}
							else {
								// e_internal += scale_1_4[idx] * (stre_vdW_Hb[myidx] + e_desolv);
								e_internal += pfloat_arraygpu[FLOATSCALE14 * num_individualsgpu + idx] * (stre_vdW_Hb[myidx] + e_desolv);
							}
						} else {
							// fprintf(stderr," stre_vdW_Hb[%d][%d][%d] = %f\n", index_1t_NEINT, t2, t1, stre_vdW_Hb[index_1t_NEINT * ATOM_MAPS * ATOM_MAPS + t2 * ATOM_MAPS + t1]);
							// e_internal += stre_vdW_Hb[index_1t_NEINT * ATOM_MAPS * ATOM_MAPS + t2 * ATOM_MAPS + t1] + e_desolv;
							myidx = index_1t_NEINT * ATOM_MAPS * ATOM_MAPS + t2 * ATOM_MAPS + t1;
							if (myidx == NEINT * ATOM_MAPS * ATOM_MAPS) {
								e_internal += stre_vdW_Hb[myidx-1] + e_desolv;
								// fprintf(stderr,"NEINT = %d, index = %d, t2 = %d, t1 = %d\n", NEINT, index_1t_NEINT, t2, t1);
							}
							else {
								e_internal += stre_vdW_Hb[myidx] + e_desolv;
							}
					
						}
						
						
					}
					total_e_internal += e_internal;
				}
				
				if (nb_group == INTRA_LIGAND) 
				{
					Nb_group_energy[INTRA_LIGAND] = total_e_internal;
				} else if (nb_group == INTER) {
					Nb_group_energy[INTER] = total_e_internal - Nb_group_energy[INTRA_LIGAND];
				} else if (nb_group == INTRA_RECEPTOR) {
					Nb_group_energy[INTRA_RECEPTOR] = total_e_internal - Nb_group_energy[INTRA_LIGAND] - Nb_group_energy[INTER];
				}
			}
			
			if(b_comp_intermolgpu) {
				//energiesgpu[idx] += ((float)total_e_internal - (float)unboundinternalFEs[idx]);
				penergiesgpu[idx] += ((float)total_e_internal - pfloat_arraygpu[FLOATUNBOUNDINTERNAL * num_individualsgpu + idx]);//(float)unboundinternalFEs[idx]);
			}
			else {
				//energiesgpu[idx] = ((float)total_e_internal - (float)unboundinternalFEs[idx]);
				penergiesgpu[idx] = ((float)total_e_internal - pfloat_arraygpu[FLOATUNBOUNDINTERNAL * num_individualsgpu + idx]);//(float)unboundinternalFEs[idx]);
			}
		}
	}
}

// const_2 uses fewer constant memory arrays (nnb and nonbondlists only)
__global__ void eintcal_kernel_const_2(unsigned int num_individualsgpu,
								int natomsgpu, 
								float *penergiesgpu, 
								float *tcoord, 
								int B_include_1_4_interactions, // Boole
								int B_have_flexible_residues, // Boole
								int *nnb_array, 
								float *Nb_group_energy, 
								float *stre_vdW_Hb, 
								float *strsol_fn, 
								float *strepsilon_fn, 
								float *strr_epsilon_fn,
								int b_comp_intermolgpu, // Boole
								float *pfloat_arraygpu,
								int *pint_arraygpu)
{
	//int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = threadIdx.x;

	if (idx + blockDim.x * blockIdx.x < num_individualsgpu) {
		
		if (!pint_arraygpu[INTEVALFLAG * num_individualsgpu + idx])//!evalflagsgpu[idx])
		{
			
# ifndef NOSQRT
			float r = 0.0f;
			// float nbc = B_use_non_bond_cutoff[idx] ? NBC : 999;
			float nbc = (int)pint_arraygpu[INTNONBONDCUT * num_individualsgpu + idx] ? NBC : 999; // Boole cast
# else
			// float nbc2 = B_use_non_bond_cutoff[idx] ? NBC2 : 999 * 999;
			float nbc2 = (int)pint_arraygpu[INTNONBONDCUT * num_individualsgpu + idx] ? NBC2 : 999 * 999; // Boole cast
# endif
			float dx = 0.0f, dy = 0.0f, dz = 0.0f;
			float r2 = 0.0f;
			
			float total_e_internal = 0.0f;
			
			float e_elec = 0.0f;
			
			int inb = 0;
			int a1 = 0, a2 = 0;
			int t1 = 0, t2 = 0;
			int nonbond_type = 0;
			
			int index_1t_NEINT = 0;
			int index_1t_NDIEL = 0;
			int nb_group = 0;
			int inb_from = 0;
			int inb_to = 0;
			int nb_group_max = 1;
			
			if (B_have_flexible_residues) {
				nb_group_max = 3;
			}
			
			for (nb_group = 0; nb_group < nb_group_max; nb_group++)
			{
				if (nb_group == 0) {
					inb_from = 0;
				} else {
					inb_from = nnb_array_c[nb_group-1];
				}
				inb_to = nnb_array_c[nb_group];
				
				for (inb = inb_from; inb < inb_to; inb++)
				{
				
					float e_internal = 0.0f;
					float e_desolv = 0.0f;
					
					a1 = (int)nonbondlist_c[inb * 7 + 0];
					a2 = (int)nonbondlist_c[inb * 7 + 1];
					t1 = (int)nonbondlist_c[inb * 7 + 2];
					t2 = (int)nonbondlist_c[inb * 7 + 3];
					
					nonbond_type = (int)nonbondlist_c[inb * 7 + 4];
					float nb_desolv = nonbondlist_c[inb * 7 + 5];
					float q1q2 = nonbondlist_c[inb * 7 + 6];
					
					dx = tcoord[idx * natomsgpu * SPACE + a1 * SPACE + X] - tcoord[idx * natomsgpu * SPACE + a2 * SPACE + X];
					dy = tcoord[idx * natomsgpu * SPACE + a1 * SPACE + Y] - tcoord[idx * natomsgpu * SPACE + a2 * SPACE + Y];
					dz = tcoord[idx * natomsgpu * SPACE + a1 * SPACE + Z] - tcoord[idx * natomsgpu * SPACE + a2 * SPACE + Z];
					
	#ifndef NOSQRT
					r = clamp(hypotenuse(dx,dy,dz), RMIN_ELEC);
					r2 = r*r;
					int index = Ang_to_index(r);
					
	#else
					r2 = sqhypotenuse(dx,dy,dz);
					r2 = clamp(r2, RMIN_ELEC2);
					int index = SqAng_to_index(r2);
	#endif
					
					index_1t_NEINT = BoundedNeint(index);
					index_1t_NDIEL = BoundedNdiel(index);
					
					if ((int)pint_arraygpu[INTINCELEC * num_individualsgpu + idx])//B_calcIntElec[idx]) // Boole cast
					{
						float r_dielectric = strr_epsilon_fn[index_1t_NDIEL];
						e_elec = q1q2 * r_dielectric;
						e_internal = e_elec;
					}
					
					if (r2 < nbc2) {
						e_desolv = strsol_fn[index_1t_NEINT] * nb_desolv;
						int myidx;
						if (B_include_1_4_interactions != 0 && nonbond_type == 4) {
							myidx = index_1t_NEINT * ATOM_MAPS * ATOM_MAPS + t2 * ATOM_MAPS + t1;
							if (myidx == NEINT * ATOM_MAPS * ATOM_MAPS) {
								// e_internal += scale_1_4[idx] * (stre_vdW_Hb[myidx-1] + e_desolv);
								e_internal += pfloat_arraygpu[FLOATSCALE14 * num_individualsgpu + idx] * (stre_vdW_Hb[myidx-1] + e_desolv);
							}
							else {
								// e_internal += scale_1_4[idx] * (stre_vdW_Hb[myidx] + e_desolv);
								e_internal += pfloat_arraygpu[FLOATSCALE14 * num_individualsgpu + idx] * (stre_vdW_Hb[myidx] + e_desolv);
							}
						} else {
							// fprintf(stderr," stre_vdW_Hb[%d][%d][%d] = %f\n", index_1t_NEINT, t2, t1, stre_vdW_Hb[index_1t_NEINT * ATOM_MAPS * ATOM_MAPS + t2 * ATOM_MAPS + t1]);
							// e_internal += stre_vdW_Hb[index_1t_NEINT * ATOM_MAPS * ATOM_MAPS + t2 * ATOM_MAPS + t1] + e_desolv;
							myidx = index_1t_NEINT * ATOM_MAPS * ATOM_MAPS + t2 * ATOM_MAPS + t1;
							if (myidx == NEINT * ATOM_MAPS * ATOM_MAPS) {
								e_internal += stre_vdW_Hb[myidx-1] + e_desolv;
								// fprintf(stderr,"NEINT = %d, index = %d, t2 = %d, t1 = %d\n", NEINT, index_1t_NEINT, t2, t1);
							}
							else {
								e_internal += stre_vdW_Hb[myidx] + e_desolv;
							}
					
						}
						
						
					}
					total_e_internal += e_internal;
				}
				
				if (nb_group == INTRA_LIGAND) 
				{
					Nb_group_energy[INTRA_LIGAND] = total_e_internal;
				} else if (nb_group == INTER) {
					Nb_group_energy[INTER] = total_e_internal - Nb_group_energy[INTRA_LIGAND];
				} else if (nb_group == INTRA_RECEPTOR) {
					Nb_group_energy[INTRA_RECEPTOR] = total_e_internal - Nb_group_energy[INTRA_LIGAND] - Nb_group_energy[INTER];
				}
			}
			
			if(b_comp_intermolgpu) {
				//energiesgpu[idx] += ((float)total_e_internal - (float)unboundinternalFEs[idx]);
				penergiesgpu[idx] += ((float)total_e_internal - pfloat_arraygpu[FLOATUNBOUNDINTERNAL * num_individualsgpu + idx]);//(float)unboundinternalFEs[idx]);
			}
			else {
				//energiesgpu[idx] = ((float)total_e_internal - (float)unboundinternalFEs[idx]);
				penergiesgpu[idx] = ((float)total_e_internal - pfloat_arraygpu[FLOATUNBOUNDINTERNAL * num_individualsgpu + idx]);//(float)unboundinternalFEs[idx]);
			}
		}
	}
}


int main()
{
	// @@ Declare variables
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float time;
	
	// CPU variables
	unsigned num_individuals = NUM_INDIVIDUALS;
	int		natomsgpu = CPUNATOMS;
	float	energiescpu[NUM_INDIVIDUALS];
	float	nonbondlist[NONBONDLISTS_SIZE];
	float	tcoord[CRDS_SIZE];
	int		B_include_1_4_interactions = 1;//INC14INTERACT; // Boole
	int		B_have_flexible_residues = 1;//HAVEFLEXRESIDUES; // Boole
	int		nnb_array[NNB_ARRAY_SIZE];
	float	nb_group_energy[NB_GROUP_ENERGY_SIZE];
	float	e_vdW_Hb[EVDWHB_SIZE];
	float	sol_fn[SOLFN_SIZE];
	float	epsilon_fn[EPSILONFN_SIZE];
	float	repsilon_fn[REPSILONFN_SIZE];
	int		b_comp_intermolgpu = 1;//B_COMP_INTERMOL; // Boole
	float	pfloat_array[FLOAT_ARRAY_SIZE];
	int 	pint_array[INT_ARRAY_SIZE];

	// Initialize arrays
	std::ifstream ifile("testinput.txt");
	// energies
	for (int i = 0; i < NUM_INDIVIDUALS; ++i) {
		if (!(ifile >> energiescpu[i])) {
			std::cerr << "File read failed at line 1 element " << i << std::endl;
			return -1;
		}
	}
	// nonbondlist
	for (int i = 0; i < NONBONDLISTS_SIZE; ++i) {
		if (!(ifile >> nonbondlist[i])) {
			std::cerr << "File read failed at line 2 element " << i << std::endl;
			return -1;
		}
	}
	// tcoord
	for (int i = 0; i < CRDS_SIZE; ++i) {
		if (!(ifile >> tcoord[i])) {
			std::cerr << "File read failed at line 3 element " << i << std::endl;
			return -1;
		}
	}
	// nnbarray
	for (int i = 0; i < NNB_ARRAY_SIZE; ++i) {
		if (!(ifile >> nnb_array[i])) {
			std::cerr << "File read failed at line 4 element " << i << std::endl;
			return -1;
		}
	}
	// nb_group_energy
	for (int i = 0; i < NB_GROUP_ENERGY_SIZE; ++i) {
		if (!(ifile >> nb_group_energy[i])) {
			std::cerr << "File read failed at line 5 element " << i << std::endl;
			return -1;
		}
	}
	// e_vdW_Hb
	for (int i = 0; i < EVDWHB_SIZE; ++i) {
		if (!(ifile >> e_vdW_Hb[i])) {
			std::cerr << "File read failed at line 6 element " << i << std::endl;
			return -1;
		}
	}
	// sol_fn
	for (int i = 0; i < SOLFN_SIZE; ++i) {
		if (!(ifile >> sol_fn[i])) {
			std::cerr << "File read failed at line 7 element " << i << std::endl;
			return -1;
		}
	}
	// epsilon_fn
	for (int i = 0; i < EPSILONFN_SIZE; ++i) {
		if (!(ifile >> epsilon_fn[i])) {
			std::cerr << "File read failed at line 8 element " << i << std::endl;
			return -1;
		}
	}
	// repsilon_fn
	for (int i = 0; i < REPSILONFN_SIZE; ++i) {
		if (!(ifile >> repsilon_fn[i])) {
			std::cerr << "File read failed at line 9 element " << i << std::endl;
			return -1;
		}
	}
	// pfloat_array
	for (int i = 0; i < FLOAT_ARRAY_SIZE; ++i) {
		if (!(ifile >> pfloat_array[i])) {
			std::cerr << "File read failed at line 10 element " << i << std::endl;
			return -1;
		}
	}
	// pint_array
	for (int i = 0; i < INT_ARRAY_SIZE; ++i) {
		if (!(ifile >> pint_array[i])) {
			std::cerr << "File read failed at line 11 element " << i << std::endl;
			return -1;
		}
	}
	ifile.close();
	
	// GPU Variables
	float	*energiesgpu;
	float	*nonbondlistgpu;
	float	*tcoordgpu;
	int		*nnb_arraygpu;
	float	*Nb_group_energygpu;
	float	*stre_vdW_Hb;
	float	*strsol_fn;
	float	*strepsilon_fn;
	float	*strr_epsilon_fn;
	float	*pfloat_arraygpu;
	int		*pint_arraygpu;
	
	//@@ Allocate GPU memory here, add hipEventRecord
	hipEventRecord(start, 0);
	hipMalloc((void**) &energiesgpu, NUM_INDIVIDUALS * sizeof(float));
	hipMalloc((void**) &nonbondlistgpu, NONBONDLISTS_SIZE * sizeof(float));
	hipMalloc((void**) &tcoordgpu, CRDS_SIZE * sizeof(float));
	hipMalloc((void**) &nnb_arraygpu, NNB_ARRAY_SIZE * sizeof(int));
	hipMalloc((void**) &Nb_group_energygpu, NB_GROUP_ENERGY_SIZE * sizeof(float));
	hipMalloc((void**) &stre_vdW_Hb, EVDWHB_SIZE * sizeof(float));
	hipMalloc((void**) &strsol_fn, SOLFN_SIZE * sizeof(float));
	hipMalloc((void**) &strepsilon_fn, EPSILONFN_SIZE * sizeof(float));
	hipMalloc((void**) &strr_epsilon_fn, REPSILONFN_SIZE * sizeof(float));
	hipMalloc((void**) &pfloat_arraygpu, FLOAT_ARRAY_SIZE * sizeof(float));
	hipMalloc((void**) &pint_arraygpu, INT_ARRAY_SIZE * sizeof(int));
	hipEventRecord(stop, 0);
	std::cerr << "Requested " << NUM_INDIVIDUALS * sizeof(float) + NONBONDLISTS_SIZE * sizeof(float) + CRDS_SIZE * sizeof(float) + NNB_ARRAY_SIZE * sizeof(int) + NB_GROUP_ENERGY_SIZE * sizeof(float) + EVDWHB_SIZE * sizeof(float) + SOLFN_SIZE * sizeof(float) + EPSILONFN_SIZE * sizeof(float) + REPSILONFN_SIZE * sizeof(float) + FLOAT_ARRAY_SIZE * sizeof(float) + INT_ARRAY_SIZE * sizeof(int) << " bytes\n";

	// Synchronize
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("Memory allocation took %f milliseconds\n", time);

	//@@ Copy memory to the GPU here
	hipEventRecord(start, 0);
	hipMemcpy(energiesgpu, energiescpu, NUM_INDIVIDUALS * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(nonbondlistgpu, nonbondlist, NONBONDLISTS_SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(tcoordgpu, tcoord, CRDS_SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(nnb_arraygpu, nnb_array, NNB_ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(Nb_group_energygpu, nb_group_energy, NB_GROUP_ENERGY_SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(stre_vdW_Hb, e_vdW_Hb, EVDWHB_SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(strsol_fn, sol_fn, SOLFN_SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(strepsilon_fn, epsilon_fn, EPSILONFN_SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(strr_epsilon_fn, repsilon_fn, REPSILONFN_SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pfloat_arraygpu, pfloat_array, FLOAT_ARRAY_SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pint_arraygpu, pint_array, INT_ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);
	// Constant memory
	hipMemcpyToSymbol(HIP_SYMBOL(nonbondlist_c), nonbondlist, NONBONDLISTS_SIZE * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(nnb_array_c), nnb_array, NNB_ARRAY_SIZE * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(strsol_fn_c), sol_fn, SOLFN_SIZE * sizeof(float));
	hipEventRecord(stop, 0);

	// Synchronize
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("Memory transfer (host->device) took %f milliseconds\n", time);

	//@@ Initialize the grid and block dimensions here
	dim3 blockSize(BLOCK_SIZE);
    dim3 gridSize((num_individuals - 1) / BLOCK_SIZE + 1);
	
	//@@ Launch the original GPU Kernel here
	hipEventRecord(start, 0);
	for (int i = 0; i < 100; ++i) {
		eintcal_kernel<<<gridSize, blockSize>>>(num_individuals,
												natomsgpu, 
												energiesgpu, 
												nonbondlistgpu, 
												tcoordgpu, 
												B_include_1_4_interactions, 
												B_have_flexible_residues, 
												nnb_arraygpu, 
												Nb_group_energygpu, 
												stre_vdW_Hb, 
												strsol_fn, 
												strepsilon_fn, 
												strr_epsilon_fn,
												b_comp_intermolgpu,
												pfloat_arraygpu,
												pint_arraygpu);
	}
	hipEventRecord(stop, 0);

	// Synchronize
	//std::cerr << hipEventSynchronize(stop) << std::endl;
	hipDeviceSynchronize();
	hipEventElapsedTime(&time, start, stop);
	printf("Original kernel execution took %f milliseconds\n", time / 100.);
	
	//@@ Launch the shared mem GPU Kernel here
	hipEventRecord(start, 0);
	for (int i = 0; i < 100; ++i) {
		eintcal_kernel_shared<<<gridSize, blockSize>>>(num_individuals,
												natomsgpu, 
												energiesgpu, 
												nonbondlistgpu, 
												tcoordgpu, 
												B_include_1_4_interactions, 
												B_have_flexible_residues, 
												nnb_arraygpu, 
												Nb_group_energygpu, 
												stre_vdW_Hb, 
												strsol_fn, 
												strepsilon_fn, 
												strr_epsilon_fn,
												b_comp_intermolgpu,
												pfloat_arraygpu,
												pint_arraygpu);
	}
	hipEventRecord(stop, 0);

	// Synchronize
	//std::cerr << hipEventSynchronize(stop) << std::endl;
	hipDeviceSynchronize();
	hipEventElapsedTime(&time, start, stop);
	printf("Shared kernel execution took %f milliseconds\n", time / 100.);
	
	//@@ Launch the first const Kernel here
	hipEventRecord(start, 0);
	for (int i = 0; i < 100; ++i) {
		eintcal_kernel_const_1<<<gridSize, blockSize>>>(num_individuals,
												natomsgpu, 
												energiesgpu, 
												tcoordgpu, 
												B_include_1_4_interactions, 
												B_have_flexible_residues, 
												Nb_group_energygpu, 
												stre_vdW_Hb, 
												strepsilon_fn, 
												strr_epsilon_fn,
												b_comp_intermolgpu,
												pfloat_arraygpu,
												pint_arraygpu);
	}
	hipEventRecord(stop, 0);

	// Synchronize
	//std::cerr << hipEventSynchronize(stop) << std::endl;
	hipDeviceSynchronize();
	hipEventElapsedTime(&time, start, stop);
	printf("Const 1 kernel execution took %f milliseconds\n", time / 100.);
	
	
	hipEventRecord(start, 0);
	for (int i = 0; i < 100; ++i) {
		eintcal_kernel_const_2<<<gridSize, blockSize>>>(num_individuals,
												natomsgpu, 
												energiesgpu, 
												tcoordgpu, 
												B_include_1_4_interactions, 
												B_have_flexible_residues, 
												nnb_arraygpu, 
												Nb_group_energygpu, 
												stre_vdW_Hb, 
												strsol_fn, 
												strepsilon_fn, 
												strr_epsilon_fn,
												b_comp_intermolgpu,
												pfloat_arraygpu,
												pint_arraygpu);
	}
	hipEventRecord(stop, 0);

	// Synchronize
	hipDeviceSynchronize();
	hipEventElapsedTime(&time, start, stop);
	printf("Constant 2 kernel execution took %f milliseconds\n", time / 100.);
	
	//@@ Copy the GPU memory back to the CPU here, add hipEventRecord
	hipEventRecord(start, 0);
	hipMemcpy(energiescpu, energiesgpu, sizeof(float) * num_individuals, hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("Memory transfer (device->host) took %f milliseconds\n", time);

	//@@ Free the GPU memory here, add hipEventRecord
	hipEventRecord(start, 0);
	hipFree(energiesgpu);
	hipFree(nonbondlistgpu);
	hipFree(tcoordgpu);
	hipFree(nnb_arraygpu);
	hipFree(Nb_group_energygpu);
	hipFree(stre_vdW_Hb);
	hipFree(strsol_fn);
	hipFree(strepsilon_fn);
	hipFree(strr_epsilon_fn);
	hipFree(pfloat_arraygpu);
	hipFree(pint_arraygpu);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("Memory deallocation took %f milliseconds\n", time);
	//@@ end hipEventRecord

	FILE *ofile = fopen("testout.txt", "w");
	fprintf(ofile, "{");
	for (int i = 0; i != NUM_INDIVIDUALS; ++i) {
		fprintf(ofile, "%f, ", energiescpu[i]);
	}
	fprintf(ofile, "}\n");
	fclose(ofile);

	return 0;
}
